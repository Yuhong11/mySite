#include <stdio.h>

int main()
{
	int dev_ct;
	hipGetDeviceCount( &dev_ct );
	
	int dev_id;
	hipGetDevice( &dev_id );

	struct hipDeviceProp_t myGPU;
	hipGetDeviceProperties( &myGPU, dev_id );	

	printf("Device Count      = %d\n", dev_ct);
	printf("Device Number     = %d\n", dev_id);
	printf("Device Name       = %s\n", myGPU.name );	
	printf("Global Memory     = %zu B = %f GB\n", myGPU.totalGlobalMem,  myGPU.totalGlobalMem/(1024.0*1024.0*1024.0) );	
	printf("Shared Mem/block  = %zu\n", myGPU.sharedMemPerBlock );	
	printf("Registers /block  = %d\n", myGPU.regsPerBlock );	
	printf("Warp Size         = %d\n", myGPU.warpSize );	
	printf("Mem Pitch         = %zu\n", myGPU.memPitch );	
	printf("Max Threads/block = %d\n", myGPU.maxThreadsPerBlock );	
	
	printf("Max Threads Dim   = (%d, %d, %d)\n", myGPU.maxThreadsDim[0],\
                                                     myGPU.maxThreadsDim[1],\
                                                     myGPU.maxThreadsDim[2]);	

	printf("Max Grid Size     = (%d, %d, %d)\n", myGPU.maxGridSize[0],\
                                                     myGPU.maxGridSize[1],\
                                                     myGPU.maxGridSize[2]);	


	printf("Total Const Mem   = %zu\n", myGPU.totalConstMem );	
	printf("Major             = %d\n", myGPU.major );	
	printf("Minor             = %d\n", myGPU.minor );	
	printf("Clock Rate        = %.2f MHz\n", myGPU.clockRate/1000.0 );

	printf("Text Alignment      = %zu\n", myGPU.textureAlignment);
        printf("Device Overlap      = %d\n", myGPU.deviceOverlap);
        printf("MultiProcessorCount = %d\n", myGPU.multiProcessorCount);
	printf("Kernel Exec Timeout Enabled = %d\n", myGPU.kernelExecTimeoutEnabled);
	printf("Integrated GPU      = %d\n", myGPU.integrated);
	printf("Can Map Host Memory = %d\n", myGPU.canMapHostMemory);
	printf("Compute Mode        = %d\n", myGPU.computeMode);
	printf("Concurrent Kernels  = %d\n", myGPU.concurrentKernels);
	printf("ECC Enabled         = %d\n", myGPU.ECCEnabled);
	printf("PCI Bus ID          = %d\n", myGPU.pciBusID);
	printf("PCI Device ID       = %d\n", myGPU.pciDeviceID);
	printf("TCC Driver          = %d\n", myGPU.tccDriver);

	return 0;
}
